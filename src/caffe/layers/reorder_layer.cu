#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/common_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReorderCOnlyForward(const int num, const int channels,
        const int height, const int width, const int* positions,
        const Dtype* in, Dtype* out) {
    for (int n = 0; n < num; n++) {
        int base = n * channels * height * width;
        for (int i = 0; i < height * width; i++) {
            int position = positions[i];
            CUDA_KERNEL_LOOP(c, channels) {
                int index_src = base + c * height * width + position;
                int index_dst = base + channels * i + c;
                out[index_dst] = in[index_src];
            }
        }
    }
}

template <typename Dtype>
void ReorderLayerCOnly<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data    = (*top)[0]->mutable_gpu_data();
  const int num      = bottom[0]->num();
  const int channels = bottom[0]->channels();
  const int *pos = reinterpret_cast<const int *>(position_ptr->cpu_data());
  const int *positions = reinterpret_cast<const int *>(position_ptr->gpu_data());
  // NOLINT_NEXT_LINE(whitespace/operators)
  CHECK_EQ(num * channels * height * width, (*top)[0]->count());
  CHECK_EQ(num * channels * height * width, bottom[0]->count());
  ReorderCOnlyForward<Dtype><<<CAFFE_GET_BLOCKS(channels), CAFFE_CUDA_NUM_THREADS>>>(
      num, channels, height, width, positions, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ReorderCOnlyBackward(const int num, const int channels,
        const int height, const int width, const int* positions,
        Dtype* in, const Dtype* out) {
    for (int n = 0; n < num; n++) {
        int base = n * channels * height * width;
        for (int i = 0; i < height * width; i++) {
            int position = positions[i];
            CUDA_KERNEL_LOOP(c, channels) {
                int index_src = base + c * height * width + position;
                int index_dst = base + channels * i + c;
                in[index_src] = out[index_dst];
            }
        }
    }
}

template <typename Dtype>
void ReorderLayerCOnly<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) { return; }
  Dtype* bottom_data    = (*bottom)[0]->mutable_gpu_diff();
  const Dtype* top_data = top[0]->gpu_diff();
  const int num      = (*bottom)[0]->num();
  const int channels = (*bottom)[0]->channels();
  const int *positions = reinterpret_cast<const int *>(position_ptr->gpu_data());
  // NOLINT_NEXT_LINE(whitespace/operators)
  CHECK_EQ(num * channels * height * width, top[0]->count());
  CHECK_EQ(num * channels * height * width, (*bottom)[0]->count());
  ReorderCOnlyBackward<Dtype><<<CAFFE_GET_BLOCKS(channels), CAFFE_CUDA_NUM_THREADS>>>(
      num, channels, height, width, positions, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_CLASS(ReorderLayer);
INSTANTIATE_CLASS(ReorderLayerCOnly);

}  // namespace caffe
